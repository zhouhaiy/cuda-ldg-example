#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

// RG*RG*MAXN must fit within mytype

#define MAXN 100000
#define RG 10
#define USECPSEC 1000000ULL
#define nTPB 256

typedef double mytype;

void conv(const mytype *A, const mytype *B, mytype* out, int N) {

    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            out[i + j] += A[i] * B[j];
}

unsigned long long dtime_usec(unsigned long long prev){
  timeval tv1;
  gettimeofday(&tv1,0);
  return ((tv1.tv_sec * USECPSEC)+tv1.tv_usec) - prev;
}

__global__ void conv_Kernel2(const mytype * A, const mytype * B, mytype *out, const int N){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx < (2*N)-1){
      mytype my_sum = 0;
      for (int i = 0; i < N; i++)
        if (((idx < N) && (i <= idx)) || ((idx >= N) && (i > (idx-N)))) {
          my_sum += __ldg(A + i)*__ldg(B + idx - i);
        }
      out[idx] = my_sum;
    }
}

__global__ void conv_Kernel1(const mytype * A, const mytype * B, mytype *out, const int N){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx < (2*N)-1){
      mytype my_sum = 0;
      for (int i = 0; i < N; i++)
        if (((idx < N) && (i <= idx)) || ((idx >= N) && (i > (idx-N)))) {
          my_sum += A[i] * B[idx-i];
        }
      out[idx] = my_sum;
    }
}


int main(int argc, char *argv[]){
  mytype *h_A, *d_A, *h_result, *d_result, *result, *h_B, *d_B, *A, *B;
  if (argc != 2) {printf("must specify N on the command line\n"); return 1;}
  int my_N = atoi(argv[1]);
  if ((my_N < 1) || (my_N > MAXN)) {printf("N out of range\n"); return 1;}
  B   = (mytype *)malloc(my_N*sizeof(mytype));
  A   = (mytype *)malloc(my_N*sizeof(mytype));
  h_A = (mytype *)malloc(my_N*sizeof(mytype));
  h_B = (mytype *)malloc(my_N*sizeof(mytype));
  h_result = (mytype *)malloc(2*my_N*sizeof(mytype));
  result   = (mytype *)malloc(2*my_N*sizeof(mytype));

  hipMalloc(&d_B, my_N*sizeof(mytype));
  hipMalloc(&d_A, my_N*sizeof(mytype));
  hipMalloc(&d_result, 2*my_N*sizeof(mytype));

  for (int i=0; i < my_N; i++){
    A[i] = rand()%RG;
    B[i] = rand()%RG;
    h_A[i] = A[i];
    h_B[i] = B[i];}

  for (int i=0; i < 2*my_N; i++){
    result[i]   = 0;
    h_result[i] = 0;}

  unsigned long long cpu_time = dtime_usec(0);
  conv(A, B, result, my_N);
  cpu_time = dtime_usec(cpu_time);

  hipMemset(d_result, 0, 2*my_N*sizeof(mytype));

  int loop = 100;
  unsigned long long k1_time = 0;
  for (int i = 0; i < loop; i++) {
    unsigned long long gpu_time = dtime_usec(0);
    hipMemcpy(d_A, h_A, my_N*sizeof(mytype), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, my_N*sizeof(mytype), hipMemcpyHostToDevice);
    conv_Kernel1<<<((2*(my_N-1))+nTPB-1)/nTPB,nTPB>>>(d_A, d_B, d_result, my_N);
    hipDeviceSynchronize();
    hipMemcpy(h_result, d_result, 2*my_N*sizeof(mytype), hipMemcpyDeviceToHost);
    gpu_time = dtime_usec(gpu_time);
    k1_time += gpu_time;
  }

  unsigned long long k2_time = 0;
  for (int i = 0; i < loop; i++) {
    unsigned long long gpu_time = dtime_usec(0);
    hipMemcpy(d_A, h_A, my_N*sizeof(mytype), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, my_N*sizeof(mytype), hipMemcpyHostToDevice);
    conv_Kernel2<<<((2*(my_N-1))+nTPB-1)/nTPB,nTPB>>>(d_A, d_B, d_result, my_N);
    hipDeviceSynchronize();
    hipMemcpy(h_result, d_result, 2*my_N*sizeof(mytype), hipMemcpyDeviceToHost);
    gpu_time = dtime_usec(gpu_time);
    k2_time += gpu_time;
  }
#if 0
  for (int i = 0; i < 2*my_N; i++) 
    if (result[i] != h_result[i]) 
    {
      printf("mismatch2 at %d, cpu: %d, gpu %d\n", i, result[i], h_result[i]); 
      return 1;
    }
#endif
  printf("Finished. conv1(without ldg) time: %ldus, conv2(with ldg) time: %ldus\n", k1_time/100, k2_time/100);
  return 0;
}
